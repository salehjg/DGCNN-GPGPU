#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "common.h"


#define BLOCK_SIZE 1024
/*
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,128  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,128  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,1,1024  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,1,1024  overaxes: 1,1,1,0;


**LA_Mean: Rank: 2  dims: 5,512,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,512,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,256,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,256,0,0      overaxes: 1,0,0,0;


**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,64   overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,128  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,20,128  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,1,1024  overaxes: 1,1,1,0;
**LA_Mean: Rank: 4  dims: 5,1024,1,1024  overaxes: 1,1,1,0;


**LA_Mean: Rank: 2  dims: 5,512,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,512,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,256,0,0      overaxes: 1,0,0,0;
**LA_Mean: Rank: 2  dims: 5,256,0,0      overaxes: 1,0,0,0;


Rank and Axes :
    -Rank=4 ==> TTTF
    -Rank=2 ==> TF
*/

extern __global__ void kernel_reduce_sum_4d_try04(
        const float * __restrict__  g_idata,
        float * __restrict__  g_buff,
        float * __restrict__  g_odata,
        const int pow_y,
        const unsigned long dim0,
        const unsigned long dim1,
        const unsigned long dim2,
        const unsigned long dim3,
        const bool overaxis0,
        const bool overaxis1,
        const bool overaxis2,
        const bool overaxis3,

        const unsigned long TGC,
        const unsigned long TGPB,
        const unsigned long SPT,
        const unsigned long TGO);


// Multiplies constant coef. into whole array element wise.
__global__ void kernel_divide_by_const_try01(
        const float * __restrict__  g_idata,
        float * __restrict__  g_odata,
        const unsigned long dim,
        const float coef) {


    unsigned long tidx = blockIdx.x * blockDim.x + threadIdx.x;

    if(tidx<dim){
        //printf("*** tidx: %ld, coef: %f \t\t g_i: %f\n",tidx,coef, g_idata[tidx]);
        g_odata[tidx] = g_idata[tidx] / coef;
        //printf("*** tidx: %ld, coef: %f \t\t g_o: %f\n",tidx,coef,  g_odata[tidx]);
    }


}



void reduce_mean_4d_try02(
        float* g_idata,
        float* g_odata,
        unsigned long dim0,
        unsigned long dim1,
        unsigned long dim2,
        unsigned long dim3,
        bool overaxis0,
        bool overaxis1,
        bool overaxis2,
        bool overaxis3){

    //hipStream_t local_stream;
    //hipStreamCreate(&local_stream);


    float* g_tempbuff;

    if( !(overaxis0 && overaxis1 && overaxis2 && !overaxis3) ) {
        printf("ERROR @reduce_sum_4d_try01 --NOT IMPLEMENTED\n"); return;
    }

    // 1. reduce_sum
    {
        unsigned long block = BLOCK_SIZE;
        unsigned long SPT, TGC, TGO, TGPB, grid, TPG;

        //Dim3 slice per thread
        SPT = 512; //cte

        //thread group offset
        TGO = dim3 * SPT;

        //thread group count
        TGC = (unsigned long) ((dim0 * dim1 * dim2 + (SPT - 1)) / SPT);

        //thread group per block
        TGPB = (unsigned long) ((BLOCK_SIZE) / dim3);
        if (TGPB % 2 && TGPB > 1) TGPB--;

        //grid size
        grid = (TGC + (TGPB - 1)) / TGPB;

        TPG = (unsigned long) dim3; //threads per group

        printf("-------------------------------------------------------\n");
        printf("KERNEL_SHAPE  : %ldx%ldx%ldx%ld\n", dim0,dim1,dim2,dim3);
        printf("KERNEL_GRID  : %ld\n", grid);
        printf("KERNEL_BLOCK : %ld\n", block);
        printf("KERNEL_SPT :   %ld\n", SPT);
        printf("KERNEL_TGO :   %ld\n", TGO);
        printf("KERNEL_TGC :   %ld\n", TGC);
        printf("KERNEL_TGPB :  %ld\n", TGPB);

        float *g_buffer;
        CHECK(hipMalloc((float **) &g_tempbuff, (dim3) * sizeof(float))); // ThreadGroupCount * ThreadsPerGroup
        //CHECK(hipMalloc((float **) &g_buffer, (TGC * TPG) * sizeof(float))); // ThreadGroupCount * ThreadsPerGroup
        CHECK(hipMemset(g_tempbuff, 0, (dim3) * sizeof(float) ));
        kernel_reduce_sum_4d_try04 << < grid, block, TGPB * TPG * sizeof(float)/*, local_stream*/ >> > (
            g_idata, /*g_buffer*/ nullptr, g_tempbuff,1,
            dim0, dim1, dim2, dim3,
            overaxis0, overaxis1, overaxis2, overaxis3,
            TGC,
            TGPB,
            SPT,
            TGO
        );
        //CHECK(hipFree(g_buffer));
    }

    CHECK(hipDeviceSynchronize());

    // 2. Multiplying (1/n) to each element of resulted tensor from step 1.
    {
        unsigned long len = dim3; //Axes combination is TTTF
        unsigned long block,grid;

        float coef = (dim0*dim1*dim2);
        printf("WRAPPER: COEF: %f\n",coef);


        block = BLOCK_SIZE;
        grid = (len + block -1 )/(block);
        kernel_divide_by_const_try01 << < grid, block, 0/*, local_stream */>> > (
                g_tempbuff, g_odata, len, coef
                );
        CHECK(hipFree(g_tempbuff));
    }

}