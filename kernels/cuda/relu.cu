
#include <hip/hip_runtime.h>
//
// Created by saleh on 10/8/18.
//

__global__ void kernel_relu(const float * __restrict__ g_idata, float * __restrict__ g_odata, unsigned long len){
    unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<len){
        g_odata[idx] = g_idata[idx]>0 ? g_idata[idx] : 0;
    }
}

void activation_relu(
        const float *g_idata,
        float *g_odata,
        unsigned long len){
    unsigned long blocksize, gridsize;
    blocksize = 256;
    gridsize = (len + blocksize -1 )/blocksize;
    kernel_relu<<<gridsize,blocksize>>>(g_idata,g_odata,len);
}