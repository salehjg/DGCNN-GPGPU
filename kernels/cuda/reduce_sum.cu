
#include <hip/hip_runtime.h>
//
// Created by saleh on 7/16/18.
//

#define DIM     128
#define SMEMDIM 4     // 128/32 = 8

// COPYRIGHT "PROFESSIONAL CUDA C PROGRAMMING - CHAPTER 5 - reduceSmemUnrollShfl"
__global__ void kernel_reduce_sum_allaxes(float *g_idata, float *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    float localSum = 0;

    if (idx + 3 * blockDim.x < n)
    {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        localSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = localSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();
    if (blockDim.x >= 64 && tid < 32) smem[tid] += smem[tid + 32];
    __syncthreads();

    // unrolling warp
    localSum = smem[tid];
    if (tid < 32)
    {
        localSum += __shfl_xor(localSum, 16);
        localSum += __shfl_xor(localSum, 8);
        localSum += __shfl_xor(localSum, 4);
        localSum += __shfl_xor(localSum, 2);
        localSum += __shfl_xor(localSum, 1);
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = localSum;
}

__global__ void reduceSmem (float *g_idata, float *g_odata, unsigned int n)
{
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

// Interleaved Pair Implementation with less divergence
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

// Interleaved Pair Implementation with less divergence
__global__ void reduceInterleavedFloat (float *g_idata, float *g_odata,
                                        unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata,
                                            unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8Float (float *g_idata, float *g_odata,
                                                 unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float b1 = g_idata[idx + 4 * blockDim.x];
        float b2 = g_idata[idx + 5 * blockDim.x];
        float b3 = g_idata[idx + 6 * blockDim.x];
        float b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

// CHAPTER 05 - reduceInteger.cu PROFESSIONAL CUDA C PROGRAMMING
__global__ void kernel_reduceSmemUnroll(float *g_idata, float *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    float tmpSum = 0;

    // boundary check
    if (idx + 4 * blockDim.x <= n)
    {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)  smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)   smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

// This is the wrapper function that will be used in cpp files!
void reduce_sum_all_axes(dim3 grid, dim3 block, float *g_idata, float *g_odata, unsigned int n){
    //reduceCompleteUnrollWarps8Float<<<grid.x,block>>>(g_idata, g_odata, n);
    kernel_reduceSmemUnroll<<<grid.x,block>>>(g_idata, g_odata, n);
}

